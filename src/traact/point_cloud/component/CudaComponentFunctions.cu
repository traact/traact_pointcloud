#include "hip/hip_runtime.h"
/** Copyright (C) 2022  Frieder Pankratz <frieder.pankratz@gmail.com> **/

#include "CudaComponentFunctions.cuh"
#include <opencv2/core/cuda/common.hpp>

namespace traact::vision {

__global__ void createPointCloud(float depth_scale,
                                 cv::cuda::PtrStepSz<uint16_t> depth,
                                 cv::cuda::PtrStepSz<cv::Vec2f> xy_table,
                                 cv::cuda::PtrStepSz<cv::Vec4f> point_cloud) {

    auto x = blockDim.x * blockIdx.x + threadIdx.x;
    auto y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= depth.cols || y >= depth.rows) {
        return;
    }

    float z = depth(y, x) / depth_scale;

    auto &point = point_cloud(y, x);
    auto &xy_factor = xy_table(y, x);

    point.val[0] = xy_factor.val[0] * z;
    point.val[1] = xy_factor.val[1] * z;
    point.val[2] = z;
    point.val[3] = 1.0f;

}

__global__ void colorPointCloud(const cv::cuda::PtrStepSz<cv::Vec4f> point_cloud,
                                const cv::cuda::PtrStepSz<cv::Vec4b> color_image,
                                const Eigen::Affine3f color_to_depth,
                                cv::cuda::PtrStepSz<cv::Vec4b> color_points,
                                const int width, const int height,
                                const float fx, const float fy,
                                const float cx, const float cy,
                                const float k1,const float k2, const float k3,const float k4, const float k5, const float k6,
                                const float p1,const float p2
                                ) {

    auto x = blockDim.x * blockIdx.x + threadIdx.x;
    auto y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= point_cloud.cols || y >= point_cloud.rows) {
        return;
    }

    auto& point_depth = point_cloud(y, x);
    auto &point_color = color_points(y,x);

    if(point_depth.val[2] < 0.01){
        point_color.val[0] = 255;
        point_color.val[1] = 0;
        point_color.val[2] = 0;
        return;
    }

    Eigen::Vector3f point =  color_to_depth * Eigen::Vector3f(point_depth.val[0], point_depth.val[1], point_depth.val[2]);

    // from k4a sdk transformation_project_internal
    float codx = 0.; // center of distortion is set to 0 for Brown Conrady model
    float cody = 0.;

    float xp = point.x() / point.z() - codx;
    float yp = point.y() / point.z() - cody; // flip on y-axis due to coordinate change image vs. opengl

    float xp2 = xp * xp;
    float yp2 = yp * yp;
    float xyp = xp * yp;
    float rs = xp2 + yp2;

    float rss = rs * rs;
    float rsc = rss * rs;
    float a = 1.f + k1 * rs + k2 * rss + k3 * rsc;
    float b = 1.f + k4 * rs + k5 * rss + k6 * rsc;
    float bi;
    if (b != 0.f)
    {
        bi = 1.f / b;
    }
    else
    {
        bi = 1.f;
    }
    float d = a * bi;

    float xp_d = xp * d;
    float yp_d = yp * d;

    float rs_2xp2 = rs + 2.f * xp2;
    float rs_2yp2 = rs + 2.f * yp2;

    xp_d += rs_2xp2 * p2 + 2.f * xyp * p1;
    yp_d += rs_2yp2 * p1 + 2.f * xyp * p2;

    float xp_d_cx = xp_d + codx;
    float yp_d_cy = yp_d + cody;

    float u = xp_d_cx * fx + cx;
    float v = yp_d_cy * fy + cy;

    int image_x = u;
    int image_y = v;
    if(image_x < 0){
        image_x = 0;
    }
    if(image_x >= color_image.cols){
        image_x = color_image.cols-1;
    }
    if(image_y < 0){
        image_y = 0;
    }
    if(image_y >= color_image.rows){
        image_y = color_image.rows-1;
    }



    const auto &image_color = color_image(image_y, image_x);
    point_color.val[0] = image_color.val[2];
    point_color.val[1] = image_color.val[1];
    point_color.val[2] = image_color.val[0];

}

void createPointCloud(const cv::cuda::GpuMat &depth,
                      const cv::cuda::GpuMat &xy_table,
                      float depth_scale,
                      cv::cuda::GpuMat &point_cloud,
                      hipStream_t stream) {

    auto width = depth.cols;
    auto height = depth.rows;

    const dim3 block(16, 16, 1);
    const dim3 grid(cv::cuda::device::divUp(width, block.x),
                    cv::cuda::device::divUp(height, block.y), 1);

    createPointCloud<<<grid, block, 0, stream>>>(depth_scale,
                                                 depth,
                                                 xy_table,
                                                 point_cloud);

}
void colorPointCloud(const cv::cuda::GpuMat &point_cloud,
                     const cv::cuda::GpuMat &color_image,
                     const CameraCalibration &calibration,
                     const Eigen::Affine3f &color_to_depth,
                     cv::cuda::GpuMat &color_points,
                     hipStream_t stream) {
    auto width = point_cloud.cols;
    auto height = point_cloud.rows;

    const dim3 block(16, 16, 1);
    const dim3 grid(cv::cuda::device::divUp(width, block.x),
                    cv::cuda::device::divUp(height, block.y), 1);


//    Eigen::Vector3f euler_angles = color_to_depth.rotation().eulerAngles(0,1,2);
//
//    Eigen::Matrix3f rotation = (Eigen::AngleAxisf( euler_angles[0], Eigen::Vector3f::UnitX())
//        * Eigen::AngleAxisf(-euler_angles[1], Eigen::Vector3f::UnitY()) // invert rotation direction around y-axis
//        * Eigen::AngleAxisf(-euler_angles[2], Eigen::Vector3f::UnitZ())).matrix(); // invert rotation direction around y-axis
//    auto out = Eigen::Quaternionf(rotation);
//
//    out = { params.translation[0] / units_per_meter,
//            -params.translation[1] / units_per_meter,
//            -params.translation[2] / units_per_meter
//    };

    colorPointCloud<<<grid, block, 0, stream>>>(point_cloud, color_image, color_to_depth, color_points,
                                                calibration.width, calibration.height,
                                                calibration.fx, calibration.fy,
                                                calibration.cx, calibration.cy,
                                                calibration.radial_distortion[0], calibration.radial_distortion[1],
                                                calibration.radial_distortion[2], calibration.radial_distortion[3],
                                                calibration.radial_distortion[4], calibration.radial_distortion[5],
                                                calibration.tangential_distortion[0], calibration.tangential_distortion[1]);

}
} // traact